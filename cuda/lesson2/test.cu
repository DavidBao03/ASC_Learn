
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
		printf("Hello world from the GPU\n");
}

int main()
{
	hello_from_gpu<<<4, 4>>>();
	hipDeviceSynchronize();

	return 0;
}
