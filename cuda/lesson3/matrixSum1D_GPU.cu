#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../tools/common.cuh"

__global__ void addFromGPU(float *A, float *B, float *C, const int N)
{
		const int bid = blockIdx.x;
		const int tid = threadIdx.x;
		const int id = tid + bid * blockDim.x;

		C[id] = A[id] + B[id];
}

void initialData(float *addr, int element)
{
		for(int i = 0; i < element; i++)
		{
				addr[i] = (float)(rand() & 0xFF) / 10.f;
		}
}

int main()
{
		setGPU();

		int iElemCount = 51200;
		size_t stBytesCount = iElemCount * sizeof(float);

		float *fpHost_A, *fpHost_B, *fpHost_C;
		fpHost_A = (float *)malloc(stBytesCount);
		fpHost_B = (float *)malloc(stBytesCount);
		fpHost_C = (float *)malloc(stBytesCount);
		
		if(fpHost_A != NULL && fpHost_B != NULL && fpHost_C != NULL)
		{	
				memset(fpHost_A, 0, stBytesCount);
				memset(fpHost_B, 0, stBytesCount);	
				memset(fpHost_C, 0, stBytesCount);
		}
		else
		{
				printf("Fail to allocate host memory!\n");
				exit(-1);
		}

		float *fpDevice_A, *fpDevice_B, *fpDevice_C;
		hipMalloc((float **)&fpDevice_A, stBytesCount);
		hipMalloc((float **)&fpDevice_B, stBytesCount);
		hipMalloc((float **)&fpDevice_C, stBytesCount);

		if(fpDevice_A != NULL && fpDevice_B != NULL && fpDevice_C != NULL)
		{
			hipMemset(fpDevice_A, 0, stBytesCount);
			hipMemset(fpDevice_B, 0, stBytesCount);
			hipMemset(fpDevice_C, 0, stBytesCount);
		}
		else
		{
				printf("Fail to allocate device memory!\n");
				free(fpHost_A);
				free(fpHost_B);
				free(fpHost_C);
				exit(-1);
		}

		srand(666);
		initialData(fpHost_A, iElemCount);
		initialData(fpHost_B, iElemCount);

		hipMemcpy(fpDevice_A, fpHost_A, stBytesCount, hipMemcpyHostToDevice);
		hipMemcpy(fpDevice_B, fpHost_B, stBytesCount, hipMemcpyHostToDevice);
		
		dim3 block(320);
		dim3 grid(iElemCount / 320);

		addFromGPU<<<grid, block>>>(fpDevice_A, fpDevice_B, fpDevice_C, iElemCount);
		hipDeviceSynchronize();

		hipMemcpy(fpHost_C, fpDevice_C, stBytesCount, hipMemcpyDeviceToHost);

		for(int i = 0; i < 32; i++)
		{
				for(int j = 0; j < 32; j++)
					printf("%.2f %.2f %.2f\n"
								,fpHost_A[j], fpHost_B[j], fpHost_C[j]);
		}

		free(fpHost_A);
		free(fpHost_B);
		free(fpHost_C);
		hipFree(fpDevice_A);
		hipFree(fpDevice_B);
		hipFree(fpDevice_C);

		hipDeviceReset();

		return 0;
}

