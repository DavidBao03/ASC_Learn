#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

// #define CHECK(call) 
// {
//     const cudaError_t error = call;
//     if (error != cudaSuccess)
//     {
//         printf("Ereor: %s:%d, ", __FILE__, __LINE__);
//         printf("code: %d, reason: %s\n", error, cudaGetErrorString(error));
//         exit(1);
//     }
// }

void checkResult(float *hostRef, float *gpuRef, const int N) {
    double epslion = 1.0E-8;
    bool match = 1;
    for (int i = 0; i < N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epslion) {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if(match) printf("Arrays match.\n");
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
    for(int idx = 0; idx < N; idx++) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C) {
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

void initialData(float *ip, int size) {
    time_t t;
    srand((unsigned int) time(&t));

    for(int i = 0; i < size; i++) { 
        ip[i] = (float) (rand() % 0xFF) / 10.0f; 
    }
}

int main(int argc, char **argv)
{
    printf("%s starting...\n", argv[0]);

    int dev = 0;
    hipSetDevice(dev);

    int nElem = 32;
    printf("Vector size %d\n", nElem);

    size_t nBtyes = nElem * sizeof(float);
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float*)malloc(nBtyes);
    h_B = (float*)malloc(nBtyes);
    hostRef = (float*)malloc(nBtyes);
    gpuRef = (float*)malloc(nBtyes);

    initialData(h_A, nElem);
    initialData(h_B, nElem);

    memset(hostRef, 0, nBtyes);
    memset(gpuRef, 0, nBtyes);

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, nBtyes);
    hipMalloc((void**)&d_B, nBtyes);
    hipMalloc((void**)&d_C, nBtyes);

    hipMemcpy(d_A, h_A, nBtyes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBtyes, hipMemcpyHostToDevice);

    dim3 block(nElem);
    dim3 grid(nElem/block.x);

    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C);
    printf("Execution configuration <<<%d, %d>>>\n", grid.x, block.x);

    hipMemcpy(gpuRef, d_C, nBtyes, hipMemcpyDeviceToHost);

    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    checkResult(hostRef, gpuRef, nElem);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return 0;
}