#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
    double epslion = 1.0E-10;
    bool match = 1;
    for (int i = 0; i < N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epslion) {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
            break;
        }
    }

    if(match) printf("Arrays match.\n");
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
    for(int idx = 0; idx < N; idx++) {
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
    int i = blockDim.x * blockIdx.x +  threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

void initialData(float *ip, int size) {
    time_t t;
    srand((unsigned int) time(&t));

    for(int i = 0; i < size; i++) { 
        ip[i] = (float) (rand() % 0xFF) / 10.0f; 
    }
}

int main(int argc, char **argv)
{
    printf("%s starting...\n", argv[0]);

    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    int nElem = 1 << 24;
    printf("Vector size %d\n", nElem);

    size_t nBtyes = nElem * sizeof(float);
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float*)malloc(nBtyes);
    h_B = (float*)malloc(nBtyes);
    hostRef = (float*)malloc(nBtyes);
    gpuRef = (float*)malloc(nBtyes);

    double iStart, iElaps;
    iStart = cpuSecond();

    initialData(h_A, nElem);
    initialData(h_B, nElem);

    iElaps = cpuSecond() - iStart;

    memset(hostRef, 0, nBtyes);
    memset(gpuRef, 0, nBtyes);

    iStart = cpuSecond();
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    iElaps = cpuSecond() - iStart;

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, nBtyes);
    hipMalloc((void**)&d_B, nBtyes);
    hipMalloc((void**)&d_C, nBtyes);

    hipMemcpy(d_A, h_A, nBtyes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBtyes, hipMemcpyHostToDevice);

    int iLen = 512; 
    dim3 block(iLen);
    dim3 grid((nElem + block.x - 1)/block.x);

    iStart = cpuSecond();
    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;
    printf("Execution configuration <<<%d, %d>>> Time elapsed %f sec\n", grid.x, block.x, iElaps);

    hipMemcpy(gpuRef, d_C, nBtyes, hipMemcpyDeviceToHost);

    checkResult(hostRef, gpuRef, nElem);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    return 0;
}