
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU()
{
    if(threadIdx.x == 5)
        printf("Hello World from GPU thread %d!\n", threadIdx.x);
}

int main()
{
    printf("Hello World from CPU!\n");

    helloFromGPU<<<1, 10>>>();
    // cudaDeviceReset();
    hipDeviceSynchronize();
    return 0;
}