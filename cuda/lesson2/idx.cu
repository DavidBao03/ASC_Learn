
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
		const int BID = blockIdx.x;
		const int TID = threadIdx.x;

		const int ID = TID + BID * blockDim.x;

		printf("Hello from block %d and thread %d, global id = %d\n", BID, TID, ID);

}

int main()
{
		hello_from_gpu<<<2,4>>>();
		hipDeviceSynchronize();

		return 0;
}
